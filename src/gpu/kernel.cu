#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void dummyKernel() {
    printf("Hello from GPU block %d, thread %d\n",
        blockIdx.x, threadIdx.x);
}

extern "C" void launchDummyKernel() {
    dummyKernel <<<2, 4 >>> ();

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr,
            "CUDA error: %s\n",
            hipGetErrorString(err));
    }
}
